
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloFromGPU() { printf("Hello World from GPU!\n"); }

int main() {
    // Launch kernel with 1 block and 1 thread
    helloFromGPU<<<1, 1>>>();

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    printf("Hello World from CPU in main function!\n");
    return 0;
}
